#include "hip/hip_runtime.h"
#include <stdint.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


__global__
void median_filter(uint8_t* input_I,
                  uint8_t* output_I,
                  int num_Rows, 
                  int num_Cols,
                  size_t pattern_size)
{
  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;

  if (px >= num_Cols || py >= num_Rows) {
      return;
  }

  uint8_t filter[25];
  
  for (int fx = 0; fx < pattern_size; fx++) {
    for (int fy = 0; fy < pattern_size; fy++) 
    {
      int imagex = px + fx - pattern_size / 2;
      int imagey = py + fy - pattern_size / 2;
      imagex = min(max(imagex, 0), num_Cols - 1);
      imagey = min(max(imagey, 0), num_Rows - 1);
      filter[fy * pattern_size + fx] = input_I[imagey * num_Cols + imagex];
    }
  }

  for (int i = 0; i < 25; i++) {
    for (int j = 0; j < 24; j++) {
      if (filter[j] > filter[j + 1]) {
        int b = filter[j];
        filter[j] = filter[j + 1];
        filter[j + 1] = b;
      }
    }
  }

  output_I[py * num_Cols + px] = filter[12];
}


int main() {

    int height;
    int width;
    int bpp;
    uint8_t* image = stbi_load("rabbit_gray.jpg", &width, &height, &bpp, 1);
    uint8_t* out_Image = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    uint8_t* dev_Image;
    uint8_t* dev_Out_Image;

    hipMalloc(&dev_Image, width * height * sizeof(uint8_t));
    hipMalloc(&dev_Out_Image, width * height * sizeof(uint8_t));

    hipMemcpy(dev_Image, image, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_Out_Image, out_Image, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);

    median_filter<<<dim3(100, 100, 1), dim3(3, 3, 1)>>>(dev_Image, dev_Out_Image, height, width, 5);
    
    hipDeviceSynchronize();
    hipMemcpy(out_Image, dev_Out_Image, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost);
    stbi_write_png("rabbit_median_filter.png", width, height, 1, out_Image, width * 1);

    free(out_Image);
    hipFree(dev_Image);
    hipFree(dev_Out_Image);
    stbi_image_free(image);

    return 0;
}